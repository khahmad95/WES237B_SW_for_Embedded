
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <string>

__global__ void myKernel(int *a){
    uint thread_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("block[%d], thread[%d]: a[%d]=%d\n", blockIdx.x, threadIdx.x, thread_global_idx, a[thread_global_idx]);
}

int main(int argc, char* argv[]){
    int a[3] = {3,4,5};
    int *dev_a;
    uint size = 3*sizeof(int);
    hipMalloc((void**)&dev_a, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

    uint b = 1; 
    uint t = 3; 
    myKernel<<<b,t>>>(dev_a);
    hipDeviceSynchronize();
    hipFree(dev_a);
    return 0;
}
